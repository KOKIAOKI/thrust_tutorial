#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

struct Add
{
  __host__ __device__ int operator()(int num)
  {
    return num + 1;
  }
};

int main(void)
{
  int vector_size = 10;
  thrust::host_vector<int> H_in(vector_size);

  // numbering
  for (int i = 0; i < vector_size; i++)
  {
    H_in[i] = i;
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  thrust::device_vector<int> D_in(H_in.size());
  thrust::device_vector<int> D_out(H_in.size());
  thrust::host_vector<int> H_out(H_in.size());
  int size = sizeof(int) * H_in.size();
  hipMemcpyAsync(thrust::raw_pointer_cast(D_in.data()), H_in.data(), size, hipMemcpyHostToDevice, stream);

  thrust::transform(thrust::cuda::par_nosync.on(stream), D_in.begin(), D_in.end(), D_out.begin(), Add());

  hipMemcpyAsync(H_out.data(), thrust::raw_pointer_cast(D_out.data()), size, hipMemcpyDeviceToHost, stream);

  for (int i = 0; i < H_out.size(); i++)
  {
    std::cout << H_out[i] << ": Hello" << std::endl;
  }

  hipStreamDestroy(stream);
  return 0;
}