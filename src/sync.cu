#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

struct Add
{
  __host__ __device__ int operator()(int num)
  {
    return num + 1;
  }
};

int main(void)
{
  int vector_size = 10;
  thrust::host_vector<int> H_in(vector_size);

  // numbering
  for (int i = 0; i < vector_size; i++)
  {
    H_in[i] = i;
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  thrust::device_vector<int> D_in = H_in;
  thrust::device_vector<int> D_out(D_in.size());

  thrust::transform(thrust::cuda::par.on(stream), D_in.begin(), D_in.end(), D_out.begin(), Add());

  thrust::device_vector<int> H_out = D_out;

  return 0;
}