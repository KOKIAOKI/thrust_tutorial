#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

int main(void)
{
  int vector_size = 10;

  hipStream_t stream;
  hipStreamCreate(&stream);

  thrust::device_vector<int> D(vector_size);
  for (int i = 0; i < vector_size; i++)
  {
    thrust::fill(thrust::cuda::par.on(stream), D.begin(), D.end(), 0);
  }

  hipStreamDestroy(stream);

  return 0;
}