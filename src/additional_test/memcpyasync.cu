#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <chrono>

int main(void)
{
  int vector_size = 10000;
  thrust::host_vector<int> host_input(vector_size);
  thrust::fill(host_input.begin(), host_input.end(), 1);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // warming up
  for (int i = 0; i < 10; i++)
  {
    void *ptr;
    hipMallocAsync(&ptr, sizeof(int) * 1024, stream);
    hipFreeAsync(ptr, stream);
    hipHostMalloc(&ptr, sizeof(int) * 1024);
    hipHostFree(ptr);
  }

  auto t1 = std::chrono::high_resolution_clock::now();
  thrust::device_vector<int> d1(vector_size);
  thrust::device_vector<int> d2(vector_size);
  thrust::device_vector<int> d3(vector_size);
  thrust::device_vector<int> d4(vector_size);
  thrust::device_vector<int> d5(vector_size);
  thrust::device_vector<int> d6(vector_size);
  thrust::device_vector<int> d7(vector_size);
  thrust::device_vector<int> d8(vector_size);
  thrust::device_vector<int> d9(vector_size);
  thrust::device_vector<int> d10(vector_size);

  thrust::host_vector<int> h1(vector_size);
  thrust::host_vector<int> h2(vector_size);
  thrust::host_vector<int> h3(vector_size);
  thrust::host_vector<int> h4(vector_size);
  thrust::host_vector<int> h5(vector_size);
  thrust::host_vector<int> h6(vector_size);
  thrust::host_vector<int> h7(vector_size);
  thrust::host_vector<int> h8(vector_size);
  thrust::host_vector<int> h9(vector_size);
  thrust::host_vector<int> h10(vector_size);

  int size = sizeof(int) * vector_size;
  hipMemcpyAsync(thrust::raw_pointer_cast(d1.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d2.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d3.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d4.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d5.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d6.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d7.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d8.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d9.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(thrust::raw_pointer_cast(d10.data()), host_input.data(), size, hipMemcpyHostToDevice, stream);

  hipMemcpyAsync(h1.data(), thrust::raw_pointer_cast(d1.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h2.data(), thrust::raw_pointer_cast(d2.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h3.data(), thrust::raw_pointer_cast(d3.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h4.data(), thrust::raw_pointer_cast(d4.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h5.data(), thrust::raw_pointer_cast(d5.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h6.data(), thrust::raw_pointer_cast(d6.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h7.data(), thrust::raw_pointer_cast(d7.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h8.data(), thrust::raw_pointer_cast(d8.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h9.data(), thrust::raw_pointer_cast(d9.data()), size, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h10.data(), thrust::raw_pointer_cast(d10.data()), size, hipMemcpyDeviceToHost, stream);

  auto t2 = std::chrono::high_resolution_clock::now();
  double time = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1e6;
  std::cout << "[test]Execution time:" << time << "[msec] " << std::endl;

  /*test*/
  // std::cout << h10[vector_size - 1] << std::endl;

  hipStreamDestroy(stream);

  return 0;
}