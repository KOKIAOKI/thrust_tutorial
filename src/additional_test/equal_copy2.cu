#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <chrono>

int main(void)
{
  int multivec_size = 10000;
  int vector_size = 10000;
  std::vector<thrust::host_vector<int>> host_multivec(multivec_size);
  for (int i = 0; i < multivec_size; i++)
  {
    thrust::host_vector<int> a(vector_size);
    thrust::fill(a.begin(), a.end(), i);
    host_multivec[i] = a;
  }

  // Create CUDA stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  // warming up
  for (int i = 0; i < 10; i++)
  {
    void *ptr;
    hipMallocAsync(&ptr, sizeof(int) * 1024, stream);
    hipFreeAsync(ptr, stream);
    hipHostMalloc(&ptr, sizeof(int) * 1024);
    hipHostFree(ptr);
  }

  std::vector<thrust::device_vector<int>> device_multivec(multivec_size);
  std::vector<thrust::host_vector<int>> host_output_multivec(multivec_size);

  auto t1 = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < multivec_size; i++)
  {
    device_multivec[i].resize(vector_size);
    host_output_multivec[i].resize(vector_size);
  }

  for (int i = 0; i < multivec_size; i++)
  {

    device_multivec[i] = host_multivec[i];
  }

  for (int i = 0; i < multivec_size; i++)
  {
    host_output_multivec[i] = device_multivec[i];
  }
  auto t2 = std::chrono::high_resolution_clock::now();
  double time = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1e6;
  std::cout << "[test]Execution time:" << time << "[msec] " << std::endl;

  /*test*/
  // std::cout << host_output_multivec[multivec_size - 1][vector_size - 1] << std::endl;

  hipStreamDestroy(stream);
  return 0;
}