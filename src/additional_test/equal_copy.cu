#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <chrono>

int main(void)
{
  int vector_size = 10000;
  thrust::host_vector<int> host_input(vector_size);
  thrust::fill(host_input.begin(), host_input.end(), 1);

  // Create CUDA stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  // warming up
  for (int i = 0; i < 10; i++)
  {
    void *ptr;
    hipMallocAsync(&ptr, sizeof(int) * 1024, stream);
    hipFreeAsync(ptr, stream);
    hipHostMalloc(&ptr, sizeof(int) * 1024);
    hipHostFree(ptr);
  }

  auto t1 = std::chrono::high_resolution_clock::now();
  thrust::device_vector<int> d1 = host_input;
  thrust::device_vector<int> d2 = host_input;
  thrust::device_vector<int> d3 = host_input;
  thrust::device_vector<int> d4 = host_input;
  thrust::device_vector<int> d5 = host_input;
  thrust::device_vector<int> d6 = host_input;
  thrust::device_vector<int> d7 = host_input;
  thrust::device_vector<int> d8 = host_input;
  thrust::device_vector<int> d9 = host_input;
  thrust::device_vector<int> d10 = host_input;

  thrust::host_vector<int> h1 = d1;
  thrust::host_vector<int> h2 = d2;
  thrust::host_vector<int> h3 = d3;
  thrust::host_vector<int> h4 = d4;
  thrust::host_vector<int> h5 = d5;
  thrust::host_vector<int> h6 = d6;
  thrust::host_vector<int> h7 = d7;
  thrust::host_vector<int> h8 = d8;
  thrust::host_vector<int> h9 = d9;
  thrust::host_vector<int> h10 = d10;

  auto t2 = std::chrono::high_resolution_clock::now();
  double time = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1e6;
  std::cout << "[test]Execution time:" << time << "[msec] " << std::endl;

  /*test*/
  // std::cout << h10[vector_size - 1] << std::endl;

  hipStreamDestroy(stream);
  return 0;
}