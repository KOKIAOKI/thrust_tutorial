#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <chrono>

int main(void)
{
  int multivec_size = 10000;
  int vector_size = 10000;
  std::vector<thrust::host_vector<int>> host_multivec(multivec_size);
  for (int i = 0; i < multivec_size; i++)
  {
    thrust::host_vector<int> a(vector_size);
    thrust::fill(a.begin(), a.end(), i);
    host_multivec[i] = a;
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  // warming up
  for (int i = 0; i < 10; i++)
  {
    void *ptr;
    hipMallocAsync(&ptr, sizeof(int) * 1024, stream);
    hipFreeAsync(ptr, stream);
    hipHostMalloc(&ptr, sizeof(int) * 1024);
    hipHostFree(ptr);
  }

  std::vector<thrust::device_vector<int>> device_multivec(multivec_size);
  std::vector<thrust::host_vector<int>> host_output_multivec(multivec_size);

  auto t1 = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < multivec_size; i++)
  {
    device_multivec[i].resize(vector_size);
    host_output_multivec[i].resize(vector_size);
  }

  int size = sizeof(int) * vector_size;
  for (int i = 0; i < multivec_size; i++)
  {
    hipMemcpyAsync(thrust::raw_pointer_cast(device_multivec[i].data()), host_multivec[i].data(), size, hipMemcpyHostToDevice, stream);
  }

  for (int i = 0; i < multivec_size; i++)
  {
    hipMemcpyAsync(host_output_multivec[i].data(), thrust::raw_pointer_cast(device_multivec[i].data()), size, hipMemcpyDeviceToHost, stream);
  }

  auto t2 = std::chrono::high_resolution_clock::now();
  double time = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1e6;
  std::cout << "[test]Execution time:" << time << "[msec] " << std::endl;

  /*test*/
  // std::cout << host_output_multivec[multivec_size - 1][vector_size - 1] << std::endl;

  hipStreamDestroy(stream);

  return 0;
}