#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

struct Add
{
  __host__ __device__ int operator()(int num)
  {
    return num + 1;
  }
};

int main(void)
{
  int vector_size = 10;
  thrust::host_vector<int> H_in(vector_size);

  // numbering
  for (int i = 0; i < vector_size; i++)
  {
    H_in[i] = i;
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  thrust::device_vector<int> D_in(H_in.size());
  thrust::device_vector<int> D_out(H_in.size());
  thrust::host_vector<int> H_out(D_out.size());
  hipMemcpyAsync(thrust::raw_pointer_cast(D_in.data()), H_in.data(), sizeof(int) * H_in.size(), hipMemcpyHostToDevice, stream);

  thrust::transform(thrust::device.on(stream), D_in.begin(), D_in.end(), D_out.begin(), Add());

  hipMemcpyAsync(H_out.data(), thrust::raw_pointer_cast(D_out.data()), sizeof(int) * H_in.size(), hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  return 0;
}