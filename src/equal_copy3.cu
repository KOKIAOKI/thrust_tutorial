#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <chrono>

int main(void)
{
  int loop_size = 10000;
  int vector_size = 10000;
  thrust::host_vector<int> host_input(vector_size);
  thrust::fill(host_input.begin(), host_input.end(), 1);

  // Create CUDA stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  // warming up
  for (int i = 0; i < 10; i++)
  {
    void *ptr;
    hipMallocAsync(&ptr, sizeof(int) * 1024, stream);
    hipFreeAsync(ptr, stream);
    hipHostMalloc(&ptr, sizeof(int) * 1024);
    hipHostFree(ptr);
  }

  std::vector<thrust::device_vector<int>> device_multivec(loop_size);
  std::vector<thrust::host_vector<int>> host_multivec(loop_size);
  // host to device
  auto h2dt1 = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < loop_size; i++)
  {
    device_multivec[i] = host_input;
  }
  auto h2dt2 = std::chrono::high_resolution_clock::now();
  double h2dtime = std::chrono::duration_cast<std::chrono::nanoseconds>(h2dt2 - h2dt1).count() / 1e6;
  std::cout << "[host to device]Execution time:" << h2dtime << "[msec] " << std::endl;

  // device to host
  auto d2ht1 = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < loop_size; i++)
  {
    host_multivec[i] = device_multivec[i];
  }
  auto d2ht2 = std::chrono::high_resolution_clock::now();
  double d2htime = std::chrono::duration_cast<std::chrono::nanoseconds>(d2ht2 - d2ht1).count() / 1e6;
  std::cout << "[host to device]Execution time:" << d2htime << "[msec] " << std::endl;

  /*test*/
  // std::cout << host_multivec[loop_size - 1][vector_size - 1] << std::endl;

  hipStreamDestroy(stream);
  return 0;
}